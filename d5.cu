
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#define BLOCKS 12
#define BLOCKSIZE 1024
//#define BSize 32
//#define QSize (BLOCKS*BLOCKSIZE)/BSize/32
#define BSize 24
#define QSize 16

struct kernel_para{
volatile int *A, *B, *C;
volatile int size;
volatile int block;
volatile int thread;
volatile int warp;
volatile int req;
volatile int funcId;
volatile int taskId;
volatile int doneHost;
int doneGPU;
};

struct kernel_para_GPU{
int warpId;
int baseId;
int queueId;
int locId;
int taskId;
int funcId;
};

typedef struct {
int contents[BSize][QSize]; // body of queue
int last[BSize]; // position of last element
}queue;

__device__ void init_queue(queue *q){
        int tid = blockDim.x*blockIdx.x+threadIdx.x;

        if(tid < BSize){
                // start form 1, since 1st warp is used in scheduling
                if(tid == 0){
                        q->last[tid] = 1;
                }else{
                        q->last[tid] = 0;
                }

                for (int i = 0; i < QSize; i++){
                        q->contents[tid][i] = 0;
                }

        }
}

__device__ void MatMul_kernel(int *A, int *B, int *C, int M_height, int M_width, int N_width, int baseTid){
#if 1
        int row = baseTid + (threadIdx.x & 0x1f);
        if(row < M_height) {
                for (int j = 0; j < N_width; j++){
                        int sum = 0;
                        for (int k = 0; k < M_width; k++){
                                int a = A[row * M_width + k];
                                int b = B[k * N_width + j];
                                sum += a * b;

                        }
                        C[row * N_width + j] = sum;

                }
        }
#endif
}

__global__ void deviceRT(volatile int *done, volatile int *totalExecTasks, volatile kernel_para_GPU *warpPool, volatile struct kernel_para *taskBuffer, struct kernel_para *taskArgs, queue *warpQ){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int warp;
	int j;
	if(tid < 32){
		if(tid < BSize)
                        init_queue(warpQ);
		warp = 0;
		j = 0;
		while(!(*done)){
			if(*done) continue;
			if(tid < BSize){
				if(taskBuffer[tid].req == 1 && !(*done)){
					warp = taskBuffer[tid].warp;
					while(warp > 0){
						if(warpQ->contents[tid][warpQ->last[tid]] == 0){
//							printf("Scheduling:%d, %d\n", taskBuffer[tid].taskId, tid);
							warpPool[tid*QSize + warpQ->last[tid]].queueId = tid;
                                                        warpPool[tid*QSize + warpQ->last[tid]].locId = warpQ->last[tid];
							warpQ->contents[tid][warpQ->last[tid]] = 1;
							warpPool[tid*QSize + warpQ->last[tid]].baseId = j*32;
							warpPool[tid*QSize + warpQ->last[tid]].taskId = taskBuffer[tid].taskId;
							warpPool[tid*QSize + warpQ->last[tid]].warpId = 1;
							__threadfence();
							warp--;
							j++;
						}
						warpQ->last[tid]++;
                                                if(warpQ->last[tid] == QSize){
                                                        if(tid == 0){
                                                                warpQ->last[tid] = 1;
                                                        }else{
                                                                warpQ->last[tid] = 0;
                                                        }
                                                }

					
					}
				
					taskBuffer[tid].req = 0;
					j = 0;
				}
			}
		}
	}else{
#if 1
		int warpIdx = (blockIdx.x*blockDim.x + threadIdx.x)/32;
		while(!(*done)){
			while(warpPool[warpIdx].warpId == 0 && !(*done));
			if(*done) return;
			switch(taskArgs[warpPool[warpIdx].taskId].funcId){
				case 1:
			//		if((threadIdx.x & 0x1f) == 0) printf("Before:%d, %d\n", warpPool[warpIdx].taskId, warpPool[warpIdx].baseId);
				MatMul_kernel((int*)taskArgs[warpPool[warpIdx].taskId].A, (int*)taskArgs[warpPool[warpIdx].taskId].B, (int*)taskArgs[warpPool[warpIdx].taskId].C, taskArgs[warpPool[warpIdx].taskId].size, taskArgs[warpPool[warpIdx].taskId].size, taskArgs[warpPool[warpIdx].taskId].size, warpPool[warpIdx].baseId);
//                                      if((threadIdx.x & 0x1f) == 0) printf("After:%p\n", taskArgs[warpPool[warpIdx].taskId].C);
				break;
                                default: {
                                	printf("kernel Type not found\n");
                                        return;
                                }
                        }

#if 1			
			if((threadIdx.x & 0x1f) == 0){
				if((atomicSub((int*)&taskArgs[warpPool[warpIdx].taskId].doneGPU,1)) ==1){
					taskArgs[warpPool[warpIdx].taskId].doneHost = 0;
					atomicAdd((int*)&totalExecTasks[0],1);
//					printf("Kernel:%d\n", *totalExecTasks);
				}
				warpPool[warpIdx].warpId = 0;
				warpQ->contents[warpPool[warpIdx].queueId][warpPool[warpIdx].locId] = 0;
				__threadfence();
			}
#endif
		}
#endif
	}
//	}
}


