#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "para.h"

struct kernel_para{
int *A, *B;
volatile int *C;
volatile int size;
volatile int block;
volatile int thread;
volatile int warp;
volatile int req;
volatile int ready;
volatile int funcId;
volatile int taskId;
volatile int doneHost;
int doneGPU;
};

struct kernel_para_GPU{
int warpId;
int baseId;
int taskId;
};

__device__ void init_queue(struct kernel_para_GPU *warpPool){
	int warpIdxx = (blockIdx.x*blockDim.x+threadIdx.x)/32;
	if((threadIdx.x) != 0){
		warpPool[warpIdxx+threadIdx.x].warpId = 0;
	}else{
		warpPool[warpIdxx+threadIdx.x].warpId = 1;
	}
	
		
}

__device__ void MatMul_kernel(int *A, int *B, int *C, int Size, int baseTid){
#if 1
        int row = baseTid + (threadIdx.x & 0x1f);
        for (int j = 0; j < Size; j++){
        	int sum = 0;
                for (int k = 0; k < Size; k++){
                	int a = A[row * Size + k];
                        int b = B[k * Size + j];
                        	sum += a * b;

                }
		C[row * Size + j] = sum;
        }
#endif
}

__device__ void VecAdd_kernel(int *A, int *B, int *C, int size, int baseTid)
{
    int i = baseTid + (threadIdx.x & 0x1f);
                //printf("In vec add with tid %d from block %d\n",i, blockIdx.x);
//                for(int j=0; j<200000; j++)
    if (i < size)
        C[i] = A[i] + B[i];
}


__global__ void deviceRT(volatile int *done, volatile int *totalExecTasks, struct kernel_para_GPU *warpPool, volatile struct kernel_para *taskBuffer, struct kernel_para *taskArgs, volatile int *exec, volatile int *totalScheTasks){
	int warpIdxx = (blockIdx.x*blockDim.x + threadIdx.x)/32;
	int warp;
	int taskbufIter;
	int base;
	int taskbufId;
	int queuebufIter;
	int queuebufId;
	// Init warp queue contents and pointers
#if 1
	if(threadIdx.x < QSize){
		init_queue(warpPool);
		warp = 0;
		taskbufIter = 0;
		queuebufIter = 0;
		base = 0;
	}
	__syncthreads();
#endif
	// scheduling in master warps
	if(threadIdx.x < 32) {
		if(threadIdx.x != 0 && threadIdx.x < (SBuf)){
			while(!(*done)){
				if(warp > 0){
					if(warpPool[queuebufId].warpId == 0){
						warpPool[queuebufId].taskId = taskBuffer[taskbufId].taskId;
						warpPool[queuebufId].baseId = base*32;
						warpPool[queuebufId].warpId = 1;
						warp--;
						base++;
						__threadfence_block();
						if(warp == 0){
							taskBuffer[taskbufId].req = 0;
                                                        base = 0;

						}
					}// End if (warpQ->contents)
				}else{
					taskbufId = (blockIdx.x*SBuf+threadIdx.x)+(taskbufIter*BSize*SBuf);
					queuebufId = (blockIdx.x*SBuf+threadIdx.x)+(queuebufIter*BSize*SBuf);

					taskbufIter++;
                                        queuebufIter++;
                                        if(taskbufIter == SRun) taskbufIter = 0;
                                        if(queuebufIter == QRun) queuebufIter = 0;

					if(taskBuffer[taskbufId].ready == 1 && !(*done)){
                                        	taskBuffer[taskbufId].ready = 0;
                                                warp = taskBuffer[taskbufId].warp;
					}
				} // end if warp > 0
			}// End while done
		}// End if(threadIdx.x< QSize)
	}//End if(threadIdx.x < 32)

#if 1
	else{
#if 1
		while(!(*exec)){
			if(*exec) return;
			if(warpPool[warpIdxx].warpId == 1 && !(*exec)){

				MatMul_kernel(taskArgs[warpPool[warpIdxx].taskId].A, taskArgs[warpPool[warpIdxx].taskId].B, (int*)taskArgs[warpPool[warpIdxx].taskId].C, taskArgs[warpPool[warpIdxx].taskId].size, warpPool[warpIdxx].baseId);

				if((threadIdx.x & 0x1f) == 0){
					if((atomicSub((int*)&taskArgs[warpPool[warpIdxx].taskId].doneGPU,1)) ==1){
                                      		taskArgs[warpPool[warpIdxx].taskId].doneHost = 0;
				//		printf("Execution:%d, %d\n", warpIdxx, warpPool[warpIdxx].taskId);
                                       		atomicAdd((int*)&totalExecTasks[blockIdx.x],1);
						//atomicAdd((int*)&totalScheTasks[0],1);
                                	}

					warpPool[warpIdxx].warpId = 0;
					__threadfence_block();
				}
			}
		}
#endif
	}// End else
#endif
}


