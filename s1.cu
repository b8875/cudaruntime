#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <unistd.h>
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>

#define BLOCKS 12
#define BLOCKSIZE 1024
#define BSize 32
#define QSize (BLOCKS*BLOCKSIZE)/BSize/32
#define DATASIZE 128
#define THREADS 128
#define N (DATASIZE*DATASIZE)
#define tasks 512

#define imin(a, b) (a<=b?a:b)

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}   

struct kernel_para{
volatile int *A, *B, *C;
volatile int size;
volatile int block;
volatile int thread;
volatile int warp;
volatile int req;
volatile int funcId;
volatile int taskId;
volatile int doneHost;
int doneGPU;
};


struct kernel_para_GPU{
int *A, *B, *C;
int size;
int warpId;
int baseId;
int queueId;
int locId;
int taskId;
int funcId;
};

struct task_arg{
volatile int doneHost;
int doneGPU;
};


typedef struct {
int contents[BSize][QSize]; // body of queue
int first[BSize]; // position of first element
int last[BSize]; // position of last element
}queue;

extern __global__ void deviceRT(volatile int *done, queue *warpQ, volatile struct kernel_para *para, struct kernel_para *taskArgs, struct kernel_para_GPU *warpPool, int totalwarps);

int ipow(int base, int exp)
{
    int result = 1;
    while (exp)
    {
        if (exp & 1)
            result *= base;
        exp >>= 1;
        base *= base;
    }

    return result;
}

int main(int argc, char** argv){
        double startTime, endTime;
        int totalwarps = ((BLOCKSIZE*BLOCKS)/32);

	hipStream_t s1;
        hipStream_t s2;
	hipStream_t s3[BSize];
	hipStream_t s4;
	checkCudaErrors(hipSetDevice(0));
        checkCudaErrors(hipDeviceReset());


	checkCudaErrors(hipStreamCreate(&s1));
        checkCudaErrors(hipStreamCreate(&s2));
	checkCudaErrors(hipStreamCreate(&s4));
	for(int i = 0; i < BSize; i++){
		checkCudaErrors(hipStreamCreate(&s3[i]));
	}

	hipEvent_t event1;
	checkCudaErrors(hipEventCreate(&event1));
	// To interrupt the runtime
	int *done, *doneDev;
	// para buffer
	struct kernel_para *paraBuffer, *paraBufferDev;
	// warp pool in device to track free warps
        struct kernel_para_GPU *warpPool, *warpPoolDev;
	// para of task
	struct kernel_para *taskArgs, *taskArgsDev;
	// warp queue
	queue *warpQ;

#if 1	
	// done flag
        checkCudaErrors(hipHostAlloc(&done, sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipMalloc(&doneDev, sizeof(int)));
#if 0	
	int *totalExecWarps, *totalExecWarpsDev;
	checkCudaErrors(hipHostAlloc(&totalExecWarps, sizeof(int), hipHostMallocDefault));
	checkCudaErrors(hipMalloc(&totalExecWarpsDev, sizeof(int)));
#endif
	// para buffer
	checkCudaErrors(hipMalloc(&paraBufferDev, BSize*sizeof(struct kernel_para)));
        checkCudaErrors(hipHostAlloc(&paraBuffer, BSize*sizeof(struct kernel_para), NULL));
	// warp Pool in device
	checkCudaErrors(hipMalloc(&warpPoolDev, totalwarps*sizeof(struct kernel_para)));
	checkCudaErrors(hipHostAlloc(&warpPool, totalwarps*sizeof(struct kernel_para), NULL));
	// warp queue
	checkCudaErrors(hipMalloc(&warpQ, sizeof(queue)));
	// para of tasks
	checkCudaErrors(hipMalloc(&taskArgsDev, tasks*sizeof(struct kernel_para)));
        checkCudaErrors(hipHostAlloc(&taskArgs, tasks*sizeof(struct kernel_para), NULL));
	// input data
	int *aDev[tasks], *bDev[tasks], *cDev[tasks];
        int *a[tasks], *b[tasks], *c[tasks];

	for(int i=0; i<tasks; i++) {
                checkCudaErrors(hipMalloc(&aDev[i], N*sizeof(int)));
                checkCudaErrors(hipMalloc(&bDev[i], N*sizeof(int)));
                checkCudaErrors(hipMalloc(&cDev[i], N*sizeof(int)));
                checkCudaErrors(hipHostAlloc(&a[i], N*sizeof(int), NULL));
                checkCudaErrors(hipHostAlloc(&b[i], N*sizeof(int), NULL));
                checkCudaErrors(hipHostAlloc(&c[i], N*sizeof(int), NULL));
        }

	for(int i = 0; i < tasks; i++){
                for(int j=0; j<N; j++) {
                        a[i][j]= (i%32)+1;
                        b[i][j]= (i%32)+1;
                        c[i][j] = 0;
                }
        }

	for(int i = 0; i < totalwarps; i++){
		warpPool[i].warpId = 0;
	}
	// Init. of para buffer
	for(int i = 0; i < BSize; i++){
                paraBuffer[i].req = 0;
        }
	*done = 0;
//	*totalExecWarps = 0;
//	checkCudaErrors(hipMemcpyAsync (totalExecWarpsDev, totalExecWarps, sizeof(int), hipMemcpyHostToDevice, s1));
        checkCudaErrors(hipMemcpyAsync (doneDev, done, sizeof(int), hipMemcpyHostToDevice, s1));
        checkCudaErrors(hipStreamSynchronize(s1));


	for(int i = 0; i < tasks; i++){
                checkCudaErrors(hipMemcpyAsync(aDev[i], a[i] , N*sizeof(int),hipMemcpyHostToDevice, s1));
                checkCudaErrors(hipMemcpyAsync(bDev[i], b[i] , N*sizeof(int),hipMemcpyHostToDevice, s1));
                checkCudaErrors(hipMemcpyAsync(cDev[i], c[i] , N*sizeof(int),hipMemcpyHostToDevice, s1));
        }

	for(int i = 0; i < tasks; i++){
		 // init. task para
		taskArgs[i].A = aDev[i];
		taskArgs[i].B = bDev[i];
		taskArgs[i].C = cDev[i];
		taskArgs[i].size = DATASIZE;
		taskArgs[i].block = 1;
		taskArgs[i].thread = THREADS;
		taskArgs[i].warp = THREADS/32;
		taskArgs[i].funcId = 1;
		taskArgs[i].taskId = i;
		taskArgs[i].req = 1;
                taskArgs[i].doneHost = 1;
                taskArgs[i].doneGPU = THREADS/32; 

	}
	checkCudaErrors(hipMemcpyAsync(paraBufferDev, paraBuffer, BSize*sizeof(struct kernel_para),hipMemcpyHostToDevice, s1));
	checkCudaErrors(hipMemcpyAsync(taskArgsDev, taskArgs, tasks*sizeof(struct kernel_para),hipMemcpyHostToDevice, s1));
	checkCudaErrors(hipMemcpyAsync(warpPoolDev, warpPool, totalwarps*sizeof(struct kernel_para_GPU),hipMemcpyHostToDevice, s1));
	checkCudaErrors(hipStreamSynchronize(s1));

	deviceRT<<<BLOCKS,BLOCKSIZE,0, s2>>>(done, warpQ, paraBufferDev, taskArgsDev, warpPoolDev, totalwarps);	

	printf("Enter task delivery\n");
	// critical section
	startTime = my_timer();
#endif	
	int j = 0;
	while(j < tasks){
#if 1
		for(int i = 0; i < BSize; i++){
			if(paraBuffer[i].req == 0){
//				printf("Host:%d, %d\n", i, j);
				paraBuffer[i].A = taskArgs[j].A;
				paraBuffer[i].B = taskArgs[j].B;
				paraBuffer[i].C = taskArgs[j].C;
				paraBuffer[i].size = taskArgs[j].size;
				
				paraBuffer[i].block = taskArgs[j].block;
				paraBuffer[i].thread = taskArgs[j].thread;
				paraBuffer[i].warp = THREADS/32;
				paraBuffer[i].funcId = taskArgs[j].funcId;
				paraBuffer[i].taskId = taskArgs[j].taskId;
				paraBuffer[i].req = taskArgs[j].req;

				checkCudaErrors(hipMemcpyAsync(&paraBufferDev[i], &paraBuffer[i] , sizeof(struct kernel_para),hipMemcpyHostToDevice, s1));
				checkCudaErrors(hipStreamSynchronize(s1));
				j++;
				if (j == tasks) break;
			}

		}
//		printf("Done scheduling %d tasks\n", j);
		if(j == tasks) break;	
		checkCudaErrors(hipMemcpyAsync(paraBuffer, paraBufferDev , BSize*sizeof(struct kernel_para),hipMemcpyDeviceToHost, s1));
		checkCudaErrors(hipStreamSynchronize(s1));
#endif
	
	}

	endTime = my_timer();
        printf("Elapsed Time1:%lf sec.\n", (endTime-startTime));

	startTime = my_timer();
	int all = 0;
	for(int i = 0; i < tasks; i++){
		while(taskArgs[i].doneHost != 0){
			checkCudaErrors(hipMemcpyAsync(&taskArgs[i], &taskArgsDev[i], sizeof(struct kernel_para),hipMemcpyDeviceToHost, s3[i%32]));
                	checkCudaErrors(hipStreamSynchronize(s3[i%32]));
			all++;
		}
		
	}
	endTime = my_timer();
        printf("Elapsed Time2:%lf sec.\n", (endTime-startTime));
	printf("Iteration:%d\n", all);

	*done = 1;
        checkCudaErrors(hipMemcpyAsync(doneDev, done , sizeof(int),hipMemcpyHostToDevice, s1));

#if 1
	// copy back results of tasks
        for(int i=0; i<tasks; i++) {
                hipMemcpyAsync (c[i], cDev[i] , N*sizeof(int),hipMemcpyDeviceToHost, s1);
        }
        hipStreamSynchronize(s1);
#endif
#if 1
        // verification
        for (int i = 0; i < tasks; i++){
                for(int j = 0; j < N; j++){
                        if(c[i][j] != DATASIZE*ipow((i%32)+1, 2)){
                                printf("Error:%d, %d\n", i, c[i][j]);
                                break;
                        }
                }
        }
#endif

//cleanup:
	// memory free
	for(int i = 0; i < tasks; i++){
		checkCudaErrors(hipHostFree(a[i]));
		checkCudaErrors(hipHostFree(b[i]));
		checkCudaErrors(hipHostFree(c[i]));
		checkCudaErrors(hipFree(aDev[i]));
		checkCudaErrors(hipFree(bDev[i]));
		checkCudaErrors(hipFree(cDev[i]));
	}
	// stream free
	checkCudaErrors(hipStreamDestroy(s1));
        checkCudaErrors(hipStreamDestroy(s2));
	checkCudaErrors(hipStreamDestroy(s4));
	for(int i = 0; i < BSize; i++){
		checkCudaErrors(hipStreamDestroy(s3[i]));
	}
	// event free
	checkCudaErrors(hipEventDestroy(event1));
	// host data free
	checkCudaErrors(hipHostFree(done));
	checkCudaErrors(hipHostFree(paraBuffer));
	checkCudaErrors(hipHostFree(taskArgs));
	checkCudaErrors(hipHostFree(warpPool));
//	checkCudaErrors(hipHostFree(totalExecWarps));

	// device data free
	checkCudaErrors(hipFree(doneDev));
	checkCudaErrors(hipFree(paraBufferDev));
	checkCudaErrors(hipFree(taskArgsDev));
	checkCudaErrors(hipFree(warpPoolDev));
        checkCudaErrors(hipFree(warpQ));
//	checkCudaErrors(hipFree(totalExecWarpsDev));

	return 0;
}
