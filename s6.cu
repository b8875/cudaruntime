#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <unistd.h>
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>

#define BLOCKS 12
#define BLOCKSIZE 1024
//#define BSize 32
//#define QSize (BLOCKS*BLOCKSIZE)/BSize/32
#define BSize 24
#define QSize 16
#define DATASIZE 32
#define THREADS 32
#define N (DATASIZE*DATASIZE)
#define tasks 1

#define imin(a, b) (a<=b?a:b)

double my_timer()
{
struct timeval time;
double _ret_val_0;
gettimeofday(( & time), 0);
_ret_val_0=(time.tv_sec+(time.tv_usec/1000000.0));
return _ret_val_0;
}

struct kernel_para{
volatile int *A, *B, *C;
volatile int size;
volatile int block;
volatile int thread;
volatile int warp;
volatile int req;
volatile int funcId;
volatile int taskId;
volatile int doneHost;
int doneGPU;
};

struct kernel_para_GPU{
int warpId;
int baseId;
int queueId;
int locId;
int taskId;
};

typedef struct {
int contents[BSize][QSize]; // body of queue
//int last[BSize]; // position of last element
}queue;

int ipow(int base, int exp)
{
    int result = 1;
    while (exp)
    {
        if (exp & 1)
            result *= base;
        exp >>= 1;
        base *= base;
    }

    return result;
}

extern __global__ void deviceRT(volatile int *done, volatile int *totalExecTasks, volatile kernel_para_GPU *warpPool, volatile struct kernel_para *taskBuffer, struct kernel_para *taskArgs, queue *warpQ);
int main(int argc, char** argv){
        double startTime, endTime;
        int totalWarps = ((BLOCKSIZE*BLOCKS)/32);
	hipSetDevice(0);
        hipDeviceReset();

        hipStream_t s1[BSize];
	hipStream_t s2;
	hipStream_t s3;
	for(int i = 0; i < BSize; i++){
	checkCudaErrors(hipStreamCreate(&s1[i]));
	}
	checkCudaErrors(hipStreamCreate(&s2));
	checkCudaErrors(hipStreamCreate(&s3));

	// To interrupt the runtime
        int *done, *doneDev;
	int *totalExecTasks, *totalExecTasksDev;
	struct kernel_para_GPU *warpPool, *warpPoolDev;
	struct kernel_para *taskArgs, *taskArgsDev;
	struct kernel_para *taskparaBuffer, *taskparaBufferDev;

	// warp queue
        queue *warpQ;

	// warp queue
        checkCudaErrors(hipMalloc(&warpQ, sizeof(queue)));

	// done flag
        checkCudaErrors(hipHostAlloc(&done, sizeof(int), hipHostMallocDefault));
        checkCudaErrors(hipMalloc(&doneDev, sizeof(int)));

	checkCudaErrors(hipHostAlloc(&warpPool, totalWarps*sizeof(struct kernel_para_GPU), hipHostMallocDefault));
        checkCudaErrors(hipMalloc(&warpPoolDev, totalWarps*sizeof(struct kernel_para_GPU)));

	checkCudaErrors(hipHostAlloc(&totalExecTasks, sizeof(int), hipHostMallocDefault));
        checkCudaErrors(hipMalloc(&totalExecTasksDev, sizeof(int)));

	checkCudaErrors(hipHostAlloc(&taskArgs, tasks*sizeof(struct kernel_para), hipHostMallocDefault));
        checkCudaErrors(hipMalloc(&taskArgsDev, tasks*sizeof(struct kernel_para)));
	
	checkCudaErrors(hipHostAlloc(&taskparaBuffer, BSize*sizeof(struct kernel_para), hipHostMallocDefault));
        checkCudaErrors(hipMalloc(&taskparaBufferDev, BSize*sizeof(struct kernel_para)));

	// input data
        int *aDev[tasks], *bDev[tasks], *cDev[tasks];
        int *a[tasks], *b[tasks], *c[tasks];

        for(int i=0; i<tasks; i++) {
                checkCudaErrors(hipMalloc(&aDev[i], N*sizeof(int)));
                checkCudaErrors(hipMalloc(&bDev[i], N*sizeof(int)));
                checkCudaErrors(hipMalloc(&cDev[i], N*sizeof(int)));
                checkCudaErrors(hipHostAlloc(&a[i], N*sizeof(int), NULL));
                checkCudaErrors(hipHostAlloc(&b[i], N*sizeof(int), NULL));
                checkCudaErrors(hipHostAlloc(&c[i], N*sizeof(int), NULL));
        }

        for(int i = 0; i < tasks; i++){
                for(int j=0; j<N; j++) {
                        a[i][j]= (i%32)+1;
                        b[i][j]= (i%32)+1;
                        c[i][j] = 0;
                }
        }


	*done = 0;
	*totalExecTasks = 0;

	for(int i = 0; i < totalWarps; i++){
		warpPool[i].warpId = 0;
	}

	for(int i = 0; i < tasks; i++){
                checkCudaErrors(hipMemcpyAsync(aDev[i], a[i] , N*sizeof(int),hipMemcpyHostToDevice, s3));
                checkCudaErrors(hipMemcpyAsync(bDev[i], b[i] , N*sizeof(int),hipMemcpyHostToDevice, s3));
                checkCudaErrors(hipMemcpyAsync(cDev[i], c[i] , N*sizeof(int),hipMemcpyHostToDevice, s3));
        }

        for(int i = 0; i < tasks; i++){
                 // init. task para
                taskArgs[i].A = aDev[i];
                taskArgs[i].B = bDev[i];
                taskArgs[i].C = cDev[i];
                taskArgs[i].size = DATASIZE;
                taskArgs[i].block = 1;
                taskArgs[i].thread = THREADS;
                taskArgs[i].warp = THREADS/32;
                taskArgs[i].funcId = 1;
                taskArgs[i].taskId = i;
                taskArgs[i].req = 1;
                taskArgs[i].doneHost = 1;
                taskArgs[i].doneGPU = THREADS/32;
//		printf("Host:%p\n", taskArgs[i].A);

        }

	for(int i = 0; i < BSize; i++){
		taskparaBuffer[i].req = 0;
	}
	checkCudaErrors(hipMemcpyAsync(doneDev, done, sizeof(int), hipMemcpyHostToDevice, s3));
	checkCudaErrors(hipMemcpyAsync(totalExecTasksDev, totalExecTasks, sizeof(int), hipMemcpyHostToDevice, s3));
	checkCudaErrors(hipMemcpyAsync(warpPoolDev, warpPool, totalWarps*sizeof(struct kernel_para_GPU), hipMemcpyHostToDevice, s3));
	checkCudaErrors(hipMemcpyAsync(taskparaBufferDev, taskparaBuffer, BSize*sizeof(struct kernel_para), hipMemcpyHostToDevice, s3));
	checkCudaErrors(hipMemcpyAsync(taskArgsDev, taskArgs, tasks*sizeof(struct kernel_para), hipMemcpyHostToDevice, s3));
	checkCudaErrors(hipStreamSynchronize(s3));
	deviceRT<<<BLOCKS,BLOCKSIZE,0, s2>>>(doneDev, totalExecTasksDev, warpPoolDev, taskparaBufferDev, taskArgsDev, warpQ);
	// para delivery
	int j = 0;
	int c1 = 0;
//	int c2 = 0;
//	int c3 = 0;
	startTime = my_timer();
	while(j < tasks){
		for(int i = 0; i < BSize; i++){
			if(taskparaBuffer[i].req == 0){
				taskparaBuffer[i].warp = THREADS/32;
				taskparaBuffer[i].req = 1;
				taskparaBuffer[i].taskId = j;
//				printf("Host:%d\n", taskparaBuffer[i].taskId);
				checkCudaErrors(hipMemcpyAsync(&taskparaBufferDev[i], &taskparaBuffer[i], sizeof(struct kernel_para), hipMemcpyHostToDevice, s3));
				j++;
//				c3 = 1;
				if(j == tasks) break;
			}
		}
	//	checkCudaErrors(hipMemcpyAsync(taskparaBufferDev, taskparaBuffer, BSize*sizeof(struct kernel_para), hipMemcpyHostToDevice, s3));
		if(j == tasks) break;
		checkCudaErrors(hipMemcpyAsync(taskparaBuffer, taskparaBufferDev, BSize*sizeof(struct kernel_para), hipMemcpyDeviceToHost, s3));
		checkCudaErrors(hipStreamSynchronize(s3));
//		if (c3 == 1){
//			c2++;
//		}
//		c3 = 0;
		c1++;
	}
	endTime = my_timer();
        printf("Elapsed Time1:%lf sec.\n", (endTime-startTime));
	printf("Iteration1:%d\n", c1);
//	printf("The avg. of tasks in one check:%d, %d\n", tasks/c2, c2);

#if 1
	int all = 0;
	startTime = my_timer();
	while(*totalExecTasks < tasks){
		checkCudaErrors(hipMemcpyAsync(totalExecTasks, totalExecTasksDev, sizeof(int), hipMemcpyDeviceToHost, s3));
		checkCudaErrors(hipStreamSynchronize(s3));
		all++;
	//	if(all > 4000) break;
	}
//	checkCudaErrors(hipStreamSynchronize(s1));
	endTime = my_timer();
        printf("Elapsed Time2:%lf sec.\n", (endTime-startTime));
	printf("Iterations:%d, %d\n", all, *totalExecTasks);
#endif
	*done = 1;
	checkCudaErrors(hipMemcpyAsync(doneDev, done, sizeof(int), hipMemcpyHostToDevice, s3));
#if 1
	  // copy back results of tasks
        for(int i=0; i<tasks; i++) {
                checkCudaErrors(hipMemcpyAsync (c[i], cDev[i] , N*sizeof(int),hipMemcpyDeviceToHost, s3));
        }
        checkCudaErrors(hipStreamSynchronize(s3));
#endif

#if 1
        // verification
        for (int i = 0; i < tasks; i++){
                for(int j = 0; j < N; j++){
                        if(c[i][j] != DATASIZE*ipow((i%32)+1, 2)){
                                printf("Error:%d, %d\n", i, c[i][j]);
                                break;
                        }
                }
        }
#endif

	for(int i = 0; i < BSize; i++){
		checkCudaErrors(hipStreamDestroy(s1[i]));
	}

	 for(int i = 0; i < tasks; i++){
                checkCudaErrors(hipHostFree(a[i]));
                checkCudaErrors(hipHostFree(b[i]));
                checkCudaErrors(hipHostFree(c[i]));
                checkCudaErrors(hipFree(aDev[i]));
                checkCudaErrors(hipFree(bDev[i]));
                checkCudaErrors(hipFree(cDev[i]));
        }

	checkCudaErrors(hipStreamDestroy(s2));
	hipStreamDestroy(s3);

	hipHostFree(done);
	hipHostFree(totalExecTasks);
	hipHostFree(warpPool);
	hipHostFree(taskArgs);
	hipHostFree(taskparaBuffer);

	hipFree(totalExecTasksDev);
	hipFree(warpPoolDev);
	hipFree(doneDev);
	hipFree(taskArgsDev);
	hipFree(taskparaBufferDev);
	checkCudaErrors(hipFree(warpQ));
	return 0;
}


